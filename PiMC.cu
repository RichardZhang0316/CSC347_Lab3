/**
 * This program implements a parallel code via a kernel function call that calculates Pi with the Monte Carlo Algorithm
 * and the hierarchical atomics strategy.
 *
 * Users are expected to enter two arguments: the executable file and the argument that corresponds to the number 
 * of “iterations” used to compute pi with the Monte Carlo algorithm
 *
 * @author Richard Zhang {zhank20@wfu.edu}
 * @date Mar.14, 2023
 * @assignment Lab 2
 * @course CSC 347
 **/

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hiprand/hiprand_kernel.h>
#include <math.h>

#define BLOCK_SIZE 1024

// kernel to initialize the random states
__global__ void setup_kernel(hiprandState *state)
{
	int index = threadIdx.x + blockDim.x*blockIdx.x;
    hiprand_init(123456789, index, 0, &state[index]);
}

// This function calculates the percentage of the points that falled inside the unit circle which is inscribed within the square
__global__ void computePi_MC_HAS(int n, hiprandState *state, int *count)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    __shared__ int cache[BLOCK_SIZE];
    cache[threadIdx.x] = 0;
    __syncthreads();

    for (int i = 0; i < n; i++) {
        // Generate random points 
        double xCoordinate = hiprand_uniform(&state[index]);
        double yCoordinate = hiprand_uniform(&state[index]);
        if (xCoordinate*xCoordinate + yCoordinate*yCoordinate <= 1.0) {
            cache[threadIdx.x]++;
        }
    }

    // reduction
	int i = blockDim.x/2;
	while(i != 0){
		if(threadIdx.x < i){
			cache[threadIdx.x] += cache[threadIdx.x + i];
		}
		i /= 2;
		__syncthreads();
	}
	// update to our global variable count
	if(threadIdx.x == 0){
		atomicAdd(count, cache[0]);
	}
}

int main(int argc, char *argv[]) {
    int iterations;

    // Determine if there are two arguments on the command line
    if (argc != 2) {
        printf("Command line arguments are not enough: %s \n", argv[0]);
        return 1;
    }

    // Convert the second argument to integer
    iterations = atoll(argv[1]);

    // Determine if the number of iteration entered by users is legitamate
    if (iterations <= 0) {
        printf("Number of iterations should not less than 1\n");
        return 2;
    }

    // Determine the size of grid and block
    int gridSize = (iterations + BLOCK_SIZE - 1) / BLOCK_SIZE;
    int blockSize = BLOCK_SIZE;

    // To generate random numbers on the device
    hiprandState *devStates;
    hipMalloc((void**)&devStates, gridSize * blockSize * sizeof(hiprandState));

    int *count;
    int *devCount;
    count = (int*)malloc(gridSize * blockSize * sizeof(int));
    hipMalloc((void**)&devCount, gridSize * blockSize * sizeof(int));
    hipMemset(devCount, 0, sizeof(int));

    // initialize all of the random states on the GPU.
    // This kernel call can also warm up the GPU, so we don't need to call computePi_MC_HAS twice
    setup_kernel<<<gridSize, blockSize>>>(devStates);
    
    hipEvent_t start, stop; /* Measure the starting time and the ending time to calculate the time spent */
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start); /* Start the timer */
    computePi_MC_HAS<<<gridSize, blockSize>>>(iterations, devStates, devCount);
    hipEventRecord(stop); /* End the timer */

    hipDeviceSynchronize();
    hipMemcpy(count, devCount, sizeof(int), hipMemcpyDeviceToHost);

    /* Output the π result and the execution time of the kernel function to the terminal */
    float total_time = 0.0;
    hipEventElapsedTime(&total_time, start, stop);

    double pi = 4.0 * (*count) / (iterations * gridSize * blockSize);
    printf("Pi: %f\n", pi);
    printf("Time costed: %f seconds\n", total_time);

    hipFree(devStates);
    hipFree(devCount);
    return 0;
}
